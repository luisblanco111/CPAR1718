#include "hip/hip_runtime.h"
/*
 * Simulacion simplificada de bombardeo de particulas de alta energia
 *
 * Computacion Paralela (Grado en Informatica)
 * 2017/2018
 *
 * (c) 2018 Arturo Gonzalez Escribano
 *
 * Modificaciones por:
 * Luis Blanco de la Cruz
 * Rubén González Ruiz
 */
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<hip/hip_runtime.h>
#include<cputils.h>

#define PI	3.14159f
#define UMBRAL	0.001f

/* Estructura para almacenar los datos de una tormenta de particulas */
typedef struct {
	int size;
	int *posval;
} Storm;



__global__ void inicializa (float *layerDevice){

	int id = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;	
	
	layerDevice[id] = 0.0f;	
}
/* ESTA FUNCION PUEDE SER MODIFICADA */
/* Funcion para actualizar una posicion de la capa */
__global__ void actualiza( float *layerDevice,int pos, float energia ) {
	/* 1. Calcular valor absoluto de la distancia entre el
		punto de impacto y el punto k de la capa */
	int id = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	int distancia = pos - id;
	if ( distancia < 0 ) distancia = - distancia;

	/* 2. El punto de impacto tiene distancia 1 */
	distancia = distancia + 1;

	/* 3. Raiz cuadrada de la distancia */
	//float atenuacion = (float)distancia*distancia;
	//float atenuacion = (float)distancia / PI;
	float atenuacion = sqrtf( (float)distancia );

	/* 4. Calcular energia atenuada */
	float energia_k = energia / atenuacion;

	/* 5. No sumar si el valor absoluto es menor que umbral */
	if ( energia_k >= UMBRAL || energia_k <= -UMBRAL ){
		layerDevice[id] = layerDevice[id] + energia_k;
	}
}


__global__ void copia(float *layerDevice, float *layer_copyDevice){

	int id = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

	layer_copyDevice[id] = layerDevice[id];
}




__global__ void relaja(float *layerDevice,float *layer_copyDevice, int layer_size){

	int id = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

	/* 4.2.2. Actualizar capa, menos los extremos, usando valores del array auxiliar */
	if(id!=0 && id!=layer_size-1)
	layerDevice[id] = ( layer_copyDevice[id-1] + layer_copyDevice[id] + layer_copyDevice[id+1] ) / 3;
}


__global__ void maximosLocales(float *layerDevice, float *maximos,int *posiciones, int layer_size){

	int id = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;
	
	/* 4.3. Localizar maximo */
	if(id>layer_size) return;
	if(id!=0 && id!=layer_size-1){
		/* Comprobar solo maximos locales */
		if ( layerDevice[id] > layerDevice[id-1] && layerDevice[id] > layerDevice[id+1] ) {
			maximos[id] = layerDevice[id];
			posiciones[id] = id;	
		}
		else{
			maximos[id]=-1.0f;
			posiciones[id]=-1;
		}
	}
	else /*if(id==0 || id==layer_size-1)*/{
		maximos[id]=-1.0f;
		posiciones[id]=-1;
	}

}




__global__ void reduceML(float *maxin, int *posin, float *maxout,int *posout, int vSize){
	int mitad = vSize/2;
	int id = (threadIdx.x + blockIdx.x * blockDim.x) + (threadIdx.y + blockIdx.y * blockDim.y) * blockDim.x * gridDim.x;

	if (id>mitad) return;
	
	if(maxin[id]<maxin[id+mitad]){
    		maxout[id] = maxin[id+mitad];
		posout[id] = posin[id+mitad];
	}
	else if(maxin[id]==maxin[id+mitad]){
		if(posin[id]>posin[id+mitad]) //si la posicion id+mitad es menor, traerla hacia delante
			posout[id]=posin[id+mitad];
	}
	else{ //maxin[id]>maxin[id+mitad] dejar como esta
	}

    	// Extra element
    	if ( vSize%2 != 0 && id == 0 ){
		if(maxin[0]<maxin[vSize-1]){
    			maxout[0] = maxin[vSize-1];
			posout[0] = posin[vSize-1];
		}
		else if(maxin[0]==maxin[vSize-1]){
			if(posin[0]>posin[vSize-1]) //si la posicion id+mitad es menor, traerla hacia delante
				posout[0]=posin[vSize-1];
		}
		else{ //maxin[id]>maxin[id+mitad] dejar como esta
		}
	}
}

/* FUNCIONES AUXILIARES: No se utilizan dentro de la medida de tiempo, dejar como estan */
/* Funcion de DEBUG: Imprimir el estado de la capa */
void debug_print(int layer_size, float *layer, int *posiciones, float *maximos, int num_storms ) {
	int i,k;
	if ( layer_size <= 35 ) {
		/* Recorrer capa */
		for( k=0; k<layer_size; k++ ) {
			/* Escribir valor del punto */
			printf("%10.4f |", layer[k] );

			/* Calcular el numero de caracteres normalizado con el maximo a 60 */
			int ticks = (int)( 60 * layer[k] / maximos[num_storms-1] );

			/* Escribir todos los caracteres menos el ultimo */
			for (i=0; i<ticks-1; i++ ) printf("o");

			/* Para maximos locales escribir ultimo caracter especial */
			if ( k>0 && k<layer_size-1 && layer[k] > layer[k-1] && layer[k] > layer[k+1] )
				printf("x");
			else
				printf("o");

			/* Si el punto es uno de los maximos especiales, annadir marca */
			for (i=0; i<num_storms; i++) 
				if ( posiciones[i] == k ) printf(" M%d", i );

			/* Fin de linea */
			printf("\n");
		}
	}
}

/*
 * Funcion: Lectura de fichero con datos de tormenta de particulas
 */
Storm read_storm_file( char *fname ) {
	FILE *fstorm = cp_abrir_fichero( fname );
	if ( fstorm == NULL ) {
		fprintf(stderr,"Error: Opening storm file %s\n", fname );
		exit( EXIT_FAILURE );
	}

	Storm storm;	
	int ok = fscanf(fstorm, "%d", &(storm.size) );
	if ( ok != 1 ) {
		fprintf(stderr,"Error: Reading size of storm file %s\n", fname );
		exit( EXIT_FAILURE );
	}

	storm.posval = (int *)malloc( sizeof(int) * storm.size * 2 );
	if ( storm.posval == NULL ) {
		fprintf(stderr,"Error: Allocating memory for storm file %s, with size %d\n", fname, storm.size );
		exit( EXIT_FAILURE );
	}
	
	int elem;
	for ( elem=0; elem<storm.size; elem++ ) {
		ok = fscanf(fstorm, "%d %d\n", 
					&(storm.posval[elem*2]),
					&(storm.posval[elem*2+1]) );
		if ( ok != 2 ) {
			fprintf(stderr,"Error: Reading element %d in storm file %s\n", elem, fname );
			exit( EXIT_FAILURE );
		}
	}
	fclose( fstorm );

	return storm;
}

/*
 * PROGRAMA PRINCIPAL
 */
int main(int argc, char *argv[]) {
	int i,j,k;

	/* 1.1. Leer argumentos */
	if (argc<3) {
		fprintf(stderr,"Usage: %s <size> <storm_1_file> [ <storm_i_file> ] ... \n", argv[0] );
		exit( EXIT_FAILURE );
	}

	int layer_size = atoi( argv[1] );
	int num_storms = argc-2;
	Storm storms[ num_storms ];

	/* 1.2. Leer datos de storms */
	for( i=2; i<argc; i++ ) 
		storms[i-2] = read_storm_file( argv[i] );

	/* 1.3. Inicializar maximos a cero */
	float maximos[ num_storms ];
	int posiciones[ num_storms ];
	for (i=0; i<num_storms; i++) {
		maximos[i] = 0.0f;
		posiciones[i] = 0;
	}

	/* 2. Inicia medida de tiempo */
	hipSetDevice(0);
	hipDeviceSynchronize();
	double ttotal = cp_Wtime();

	/* COMIENZO: No optimizar/paralelizar el main por encima de este punto */

	/*Calculo del tamaño de bloque y de grid*/
	int tamBlock = 256;
	int tamGrid;
	
	if (layer_size % tamBlock == 0) {
		tamGrid = layer_size/tamBlock;
	} else {
		tamGrid = (layer_size/tamBlock)+1;
	}

	/*Creacion y reserva de memoria de la matrices en Device*/

	float *layerDevice;
	float *layer_copyDevice;
	float *maxDevice;
	int *posDevice;

	hipMalloc((void**) &layerDevice, sizeof(float) * layer_size);
	hipMalloc((void**) &layer_copyDevice, sizeof(float) * layer_size);
	hipMalloc((void**) &maxDevice, sizeof(float) * layer_size);
	hipMalloc((void**) &posDevice, sizeof(int) * layer_size);
	
	float *maxAbs=(float *)malloc( sizeof(float));
	int *posAbs=(int *)malloc( sizeof(int));

	inicializa<<<tamGrid, tamBlock>>>(layerDevice);
	/* 4. Fase de bombardeos */
	for( i=0; i<num_storms; i++) {
	/*Pasar de Host a Device */	
	

		/* 4.1. Suma energia de impactos */
		/* Para cada particula */
		for( j=0; j<storms[i].size; j++ ) {
			/* Energia de impacto (en milesimas) */
			float energia = (float)storms[i].posval[j*2+1] / 1000;
			/* Posicion de impacto */
			int posicion = storms[i].posval[j*2];

			/* Para cada posicion de la capa */
				/* Actualizar posicion */
			actualiza<<<tamGrid,tamBlock>>>(layerDevice,posicion, energia);
		}

		copia<<<tamGrid, tamBlock>>>(layerDevice, layer_copyDevice);
		relaja<<<tamGrid,tamBlock>>>(layerDevice,layer_copyDevice,layer_size);
		
		maximosLocales<<<tamGrid,tamBlock>>>(layerDevice, maxDevice, posDevice, layer_size);

		for( k=layer_size; k>1; k/=2 )	
		reduceML<<<tamGrid,tamBlock>>>(maxDevice,posDevice,maxDevice,posDevice,k);
	
		hipMemcpy(maxAbs,&maxDevice[0], sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(posAbs,&posDevice[0], sizeof(int),hipMemcpyDeviceToHost);
	
		maximos[i]=maxAbs[0];
		posiciones[i]=posAbs[0];
	
	}


	/* FINAL: No optimizar/paralelizar por debajo de este punto */

	/* 6. Final de medida de tiempo */
	hipDeviceSynchronize();
	ttotal = cp_Wtime() - ttotal;

	/* 7. DEBUG: Dibujar resultado (Solo para capas con hasta 35 puntos) */
	#ifdef DEBUG
	debug_print( layer_size, layer, posiciones, maximos, num_storms );
	#endif

	/* 8. Salida de resultados para tablon */
	printf("\n");
	/* 8.1. Tiempo total de la computacion */
	printf("Time: %lf\n", ttotal );
	/* 8.2. Escribir los maximos */
	printf("Result:");
	for (i=0; i<num_storms; i++)
		printf(" %d %f", posiciones[i], maximos[i] );
	printf("\n");

	/* 9. Liberar recursos */	
	for( i=0; i<argc-2; i++ )
		free( storms[i].posval );

	/* 10. Final correcto */
	return 0;
}
